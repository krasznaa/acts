#include "hip/hip_runtime.h"
// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

// CUDA plugin include(s).
#include "Acts/Plugins/Cuda/Seeding/FindTriplets.hpp"
#include "Acts/Plugins/Cuda/Seeding/Types.hpp"
#include "Acts/Plugins/Cuda/Utilities/DeviceMatrix.cuh"
#include "Acts/Plugins/Cuda/Utilities/ErrorCheck.cuh"
#include "Acts/Plugins/Cuda/Utilities/HostMatrix.hpp"

// CUDA include(s).
#include <hip/hip_runtime.h>

// System include(s).
#include <cmath>

#include <iostream>

namespace Acts {
namespace Cuda {
namespace kernels {

__global__ void findTriplets(int nTripletCandidates,
                             std::size_t nBottomSP, const float* bottomSPArray,
                             std::size_t nMiddleSP, const float* middleSPArray,
                             std::size_t nTopSP, const float* topSPArray,
                             const int* middleBottomCountArray,
                             const int* middleBottomArray,
                             const int* middleTopCountArray,
                             const int* middleTopArray) {

  // Get the global index.
  const int tripletIndex = blockIdx.x * blockDim.x + threadIdx.x;

  // If we're out of bounds, finish right away.
  if (tripletIndex >= nTripletCandidates) {
    return;
  }

  // Create helper objects on top of the spacepoint arrays.
  const std::size_t bottomSPSize[] = {nBottomSP, details::SP_DIMENSIONS};
  DeviceMatrix<2, float> bottomSPs(bottomSPSize, bottomSPArray);
  const std::size_t middleSPSize[] = {nMiddleSP, details::SP_DIMENSIONS};
  DeviceMatrix<2, float> middleSPs(middleSPSize, middleSPArray);
  const std::size_t topSPSize[] = {nTopSP, details::SP_DIMENSIONS};
  DeviceMatrix<2, float> topSPs(topSPSize, topSPArray);

  // Create helper objects on top of the dublet matrices.
  const std::size_t middleBottomMatrixSize[] = {nMiddleSP, nBottomSP};
  DeviceMatrix<2, int> middleBottomMatrix(middleBottomMatrixSize,
                                          middleBottomArray);
  const std::size_t middleTopMatrixSize[] = {nMiddleSP, nTopSP};
  DeviceMatrix<2, int> middleTopMatrix(middleTopMatrixSize,
                                       middleTopArray);

  // Find the dublet pair to evaluate.
  std::size_t middleIndex = 0;
  int runningIndex = tripletIndex;
  int tmpValue = 0;
  while (runningIndex >= (tmpValue = (middleBottomCountArray[middleIndex] *
                                      middleTopCountArray[middleIndex]))) {
    assert(middleIndex < nMiddleSP);
    middleIndex += 1;
    runningIndex -= tmpValue;
  }
  std::size_t bottomMatrixIndex =
    runningIndex / middleTopCountArray[middleIndex];
  assert(bottomMatrixIndex < middleBottomCountArray[middleIndex]);
  std::size_t topMatrixIndex = runningIndex % middleTopCountArray[middleIndex];
  std::size_t middleBottomMatrixIndex[] = {middleIndex, bottomMatrixIndex};
  std::size_t middleTopMatrixIndex[] = {middleIndex, topMatrixIndex};
  std::size_t bottomIndex = middleBottomMatrix.get(middleBottomMatrixIndex);
  assert(bottomIndex < nBottomSP);
  std::size_t topIndex = middleTopMatrix.get(middleTopMatrixIndex);
  assert(topIndex < nTopSP);

  // Extract the properties of the selected spacepoints.
  std::size_t middleXIndex[] = {middleIndex, details::SP_X_INDEX};
  std::size_t middleYIndex[] = {middleIndex, details::SP_Y_INDEX};
  std::size_t middleZIndex[] = {middleIndex, details::SP_Z_INDEX};
  std::size_t middleRIndex[] = {middleIndex, details::SP_R_INDEX};
  std::size_t middleVZIndex[] = {middleIndex, details::SP_VZ_INDEX};
  std::size_t middleVRIndex[] = {middleIndex, details::SP_VR_INDEX};

  float xM = middleSPs.get(middleXIndex);
  float yM = middleSPs.get(middleYIndex);
  float zM = middleSPs.get(middleZIndex);
  float rM = middleSPs.get(middleRIndex);
  float varianceZM = middleSPs.get(middleVZIndex);
  float varianceRM = middleSPs.get(middleVRIndex);
  float cosPhiM = xM / rM;
  float sinPhiM = yM / rM;

  return;
}

}  // namespace kernels

namespace details {

void findTriplets(int maxBlockSize, const DubletCounts& dubletCounts,
                  std::size_t nBottomSP,
                  const device_array<float>& bottomSPDeviceMatrix,
                  std::size_t nMiddleSP,
                  const device_array<float>& middleSPDeviceMatrix,
                  std::size_t nTopSP,
                  const device_array<float>& topSPDeviceMatrix,
                  const device_array<int>& middleBottomCountArray,
                  const device_array<int>& middleBottomArray,
                  const device_array<int>& middleTopCountArray,
                  const device_array<int>& middleTopArray) {

  // Calculate the parallelisation for the triplet finding.
  const int numBlocks =
      (dubletCounts.nTriplets + maxBlockSize - 1) / maxBlockSize;

  // Launch the triplet finding.
  kernels::findTriplets<<<numBlocks, maxBlockSize>>>(
      dubletCounts.nTriplets, nBottomSP, bottomSPDeviceMatrix.get(), nMiddleSP,
      middleSPDeviceMatrix.get(), nTopSP, topSPDeviceMatrix.get(),
      middleBottomCountArray.get(), middleBottomArray.get(),
      middleTopCountArray.get(), middleTopArray.get());
  ACTS_CUDA_ERROR_CHECK(hipGetLastError());
  ACTS_CUDA_ERROR_CHECK(hipDeviceSynchronize());
  return;
}

}  // namespace details
}  // namespace Cuda
}  // namespace Acts
