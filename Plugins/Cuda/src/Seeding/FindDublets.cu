#include "hip/hip_runtime.h"
// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

// CUDA plugin include(s).
#include "Acts/Plugins/Cuda/Seeding/FindDublets.hpp"
#include "Acts/Plugins/Cuda/Seeding/Types.hpp"
#include "Acts/Plugins/Cuda/Utilities/ErrorCheck.cuh"
#include "Acts/Plugins/Cuda/Utilities/MatrixMacros.hpp"

// CUDA include(s).
#include <hip/hip_runtime.h>

// System include(s).
#include <cassert>
#include <cmath>

namespace {

/// Type of "other spacepoint" passed to the kernel
enum OtherSPType : int {
  BottomSP = 0, //< The "other" spacepoint is a bottom one
  TopSP = 1 //< The "other" spacepoint is a top one
};

} // private namespace

namespace Acts {
namespace Cuda {
namespace kernels {

template<int SPType>
__device__ float getDeltaR(float /*middleR*/, float /*otherR*/) {
  // This function should *never* be called.
  assert(false);
  return 0.0f;
}

template<>
__device__ float getDeltaR<BottomSP>(float middleR, float bottomR) {
  return middleR - bottomR;
}

template<>
__device__ float getDeltaR<TopSP>(float middleR, float topR) {
  return topR - middleR;
}

template<int SPType>
__device__ float getCotTheta(float /*middleZ*/, float /*otherZ*/,
                             float /*deltaR*/) {
  // This function should *never* be called.
  assert(false);
  return 0.0f;
}

template<>
__device__ float getCotTheta<BottomSP>(float middleZ, float bottomZ,
                                       float deltaR) {
  return (middleZ - bottomZ) / deltaR;
}

template<>
__device__ float getCotTheta<TopSP>(float middleZ, float topZ, float deltaR) {
  return (topZ - middleZ) / deltaR;
}

template<int SPType>
__global__ void findDublets(std::size_t nMiddleSP,
                            const details::SpacePoint* middleSPArray,
                            std::size_t nOtherSP,
                            const details::SpacePoint* otherSPArray,
                            float deltaRMin, float deltaRMax, float cotThetaMax,
                            float collisionRegionMin, float collisionRegionMax,
                            int* compCountArray, int* compArray) {

  // Figure out which dublet the kernel operates on.
  const std::size_t middleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  const std::size_t otherIndex = blockIdx.y * blockDim.y + threadIdx.y;

  // If we're outside of bounds, stop here.
  if ((middleIndex >= nMiddleSP) || (otherIndex >= nOtherSP)) {
    return;
  }

  // Calculate variables used in the compatibility check.
  const float deltaR = getDeltaR<SPType>(middleSPArray[middleIndex].radius,
                                         otherSPArray[otherIndex].radius);
  const float cotTheta = getCotTheta<SPType>(middleSPArray[middleIndex].z,
                                             otherSPArray[otherIndex].z,
                                             deltaR);
  const float zOrigin = middleSPArray[middleIndex].z -
      middleSPArray[middleIndex].radius * cotTheta;

  // Perform the compatibility check.
  const bool isCompatible = ((deltaR >= deltaRMin) && (deltaR <= deltaRMax) &&
                             (fabs(cotTheta) <= cotThetaMax) &&
                             (zOrigin >= collisionRegionMin) &&
                             (zOrigin <= collisionRegionMax));

  // If they are compatible, save their indices into the output matrix.
  if (isCompatible) {
    const int compRow = atomicAdd(compCountArray + middleIndex, 1);
    ACTS_CUDA_MATRIX2D_ELEMENT(compArray, nMiddleSP, nOtherSP, middleIndex,
                               compRow) = otherIndex;
  }
  return;
}

__global__ void countDublets(std::size_t nMiddleSP,
                             const int* middleBottomCountArray,
                             const int* middleTopCountArray,
                             details::DubletCounts* dubletCounts) {

  extern __shared__ details::DubletCounts sum[];

  // Get the thread identifier. Note that the kernel launch requests half as
  // many threads than how many elements we have in the arrays.
  const int middleIndex = blockIdx.x * blockDim.x * 2 + threadIdx.x;

  details::DubletCounts thisSum;
  if (middleIndex < nMiddleSP) {
    thisSum.nDublets = (middleBottomCountArray[middleIndex] +
                        middleTopCountArray[middleIndex]);
    thisSum.nTriplets = (middleBottomCountArray[middleIndex] *
                         middleTopCountArray[middleIndex]);
    thisSum.maxMBDublets = middleBottomCountArray[middleIndex];
    thisSum.maxMTDublets = middleTopCountArray[middleIndex];
    thisSum.maxTriplets = thisSum.nTriplets;
  }
  if (middleIndex + blockDim.x < nMiddleSP) {
    thisSum.nDublets += (middleBottomCountArray[middleIndex + blockDim.x] +
                         middleTopCountArray[middleIndex + blockDim.x]);
    thisSum.nTriplets += (middleBottomCountArray[middleIndex + blockDim.x] *
                          middleTopCountArray[middleIndex + blockDim.x]);
    thisSum.maxMBDublets = max(middleBottomCountArray[middleIndex + blockDim.x],
                               thisSum.maxMBDublets);
    thisSum.maxMTDublets = max(middleTopCountArray[middleIndex + blockDim.x],
                               thisSum.maxMTDublets);
    thisSum.maxTriplets =
        max((middleBottomCountArray[middleIndex + blockDim.x] *
             middleTopCountArray[middleIndex + blockDim.x]),
        thisSum.maxTriplets);
  }

  // Load the first sum step into shared memory.
  sum[threadIdx.x] = thisSum;
  __syncthreads();

  // Do the summation in some iterations.
  for (unsigned int i = blockDim.x / 2; i > 0; i>>=1) {
    if (threadIdx.x < i) {
      const details::DubletCounts& otherSum = sum[threadIdx.x + i];
      thisSum.nDublets += otherSum.nDublets;
      thisSum.nTriplets += otherSum.nTriplets;
      thisSum.maxMBDublets = max(thisSum.maxMBDublets, otherSum.maxMBDublets);
      thisSum.maxMTDublets = max(thisSum.maxMTDublets, otherSum.maxMTDublets);
      thisSum.maxTriplets = max(thisSum.maxTriplets, otherSum.maxTriplets);
      sum[threadIdx.x] = thisSum;
    }
    __syncthreads();
  }

  // Write the result of this execution block into the global memory.
  if (threadIdx.x == 0) {
    dubletCounts[blockIdx.x] = thisSum;
  }
  return;
}

}  // namespace kernels

namespace details {

void findDublets(std::size_t maxBlockSize,
                 std::size_t nBottomSP,
                 const device_array<SpacePoint>& bottomSPArray,
                 std::size_t nMiddleSP,
                 const device_array<SpacePoint>& middleSPArray,
                 std::size_t nTopSP,
                 const device_array<SpacePoint>& topSPArray,
                 float deltaRMin, float deltaRMax,
                 float cotThetaMax, float collisionRegionMin,
                 float collisionRegionMax,
                 device_array<int>& middleBottomCountArray,
                 device_array<int>& middleBottomArray,
                 device_array<int>& middleTopCountArray,
                 device_array<int>& middleTopArray) {

  // Calculate the parallelisation for the middle<->bottom spacepoint
  // compatibility flagging.
  const dim3 blockSizeMB(1, maxBlockSize);
  const dim3 numBlocksMB((nMiddleSP + blockSizeMB.x - 1)/blockSizeMB.x,
                         (nBottomSP + blockSizeMB.y - 1)/blockSizeMB.y);

  // Launch the middle-bottom dublet finding.
  kernels::findDublets<BottomSP><<<numBlocksMB, blockSizeMB>>>(
      nMiddleSP, middleSPArray.get(),
      nBottomSP, bottomSPArray.get(),
      deltaRMin, deltaRMax, cotThetaMax, collisionRegionMin, collisionRegionMax,
      middleBottomCountArray.get(), middleBottomArray.get());
  ACTS_CUDA_ERROR_CHECK(hipGetLastError());

  // Calculate the parallelisation for the middle<->top spacepoint
  // compatibility flagging.
  const dim3 blockSizeMT(1, maxBlockSize);
  const dim3 numBlocksMT((nMiddleSP + blockSizeMT.x - 1)/blockSizeMT.x,
                         (nTopSP    + blockSizeMT.y - 1)/blockSizeMT.y);

  // Launch the middle-bottom dublet finding.
  kernels::findDublets<TopSP><<<numBlocksMT, blockSizeMT>>>(
      nMiddleSP, middleSPArray.get(),
      nTopSP, topSPArray.get(),
      deltaRMin, deltaRMax, cotThetaMax, collisionRegionMin, collisionRegionMax,
      middleTopCountArray.get(), middleTopArray.get());
  ACTS_CUDA_ERROR_CHECK(hipGetLastError());
  ACTS_CUDA_ERROR_CHECK(hipDeviceSynchronize());
  return;
}

DubletCounts countDublets(std::size_t maxBlockSize, std::size_t nMiddleSP,
                          const device_array<int>& middleBottomCountArray,
                          const device_array<int>& middleTopCountArray) {

  // Calculate the parallelisation for the dublet counting.
  const int numBlocks = (nMiddleSP + maxBlockSize - 1) / maxBlockSize;
  const int sharedMem = maxBlockSize * sizeof(DubletCounts);

  // Create the small memory block in which we will get the count back for each
  // execution block.
  auto dubletCountsDevice = make_device_array<DubletCounts>(numBlocks);

  // Run the reduction kernel.
  kernels::countDublets<<<numBlocks, maxBlockSize, sharedMem>>>(
      nMiddleSP, middleBottomCountArray.get(), middleTopCountArray.get(),
      dubletCountsDevice.get());
  ACTS_CUDA_ERROR_CHECK(hipGetLastError());
  ACTS_CUDA_ERROR_CHECK(hipDeviceSynchronize());

  // Copy the sum(s) back to the host.
  auto dubletCountsHost = make_host_array<DubletCounts>(numBlocks);
  ACTS_CUDA_ERROR_CHECK(hipMemcpy(dubletCountsHost.get(),
                                   dubletCountsDevice.get(),
                                   numBlocks * sizeof(DubletCounts),
                                   hipMemcpyDeviceToHost));

  // Perform the final summation on the host. (Assuming that the number of
  // middle space points is not so large that it would make sense to do the
  // summation iteratively on the device.)
  DubletCounts result;
  for (int i = 0; i < numBlocks; ++i) {
    result.nDublets += dubletCountsHost.get()[i].nDublets;
    result.nTriplets += dubletCountsHost.get()[i].nTriplets;
    result.maxMBDublets = std::max(dubletCountsHost.get()[i].maxMBDublets,
                                   result.maxMBDublets);
    result.maxMTDublets = std::max(dubletCountsHost.get()[i].maxMTDublets,
                                   result.maxMTDublets);
    result.maxTriplets = std::max(dubletCountsHost.get()[i].maxTriplets,
                                  result.maxTriplets);
  }
  return result;
}

}  // namespace details
}  // namespace Cuda
}  // namespace Acts
